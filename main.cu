#include "hip/hip_runtime.h"
#include <mpi.h>
__global__ void printHelloGPU()
{
	printf("Hello x: #%d  y: #%d\n", threadIdx.x, threadIdx.y);
}        
int main(int argc, char *argv[]) {
    int rank,size;
    MPI_Init(&argc,&argv);
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    MPI_Comm_size(MPI_COMM_WORLD,&size);
	dim3 threads(3, 3);
	printHelloGPU<<<1, threads>>>();
	hipDeviceSynchronize();
    MPI_Finalize();
    return 0;
}

